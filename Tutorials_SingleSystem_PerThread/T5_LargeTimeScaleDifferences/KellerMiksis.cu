#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <sstream>
#include <fstream>

#include "KellerMiksis_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

#define PI 3.14159265358979323846

using namespace std;

// Physical control parameters
const int NumberOfFrequency = 46080;
	
// Solver Configuration
#define SOLVER RKCK45     // RK4, RKCK45
#define PRECISION double  // float, double
const int NT   = NumberOfFrequency; // NumberOfThreads
const int SD   = 2;     // SystemDimension
const int NCP  = 13;    // NumberOfControlParameters
const int NSP  = 0;     // NumberOfSharedParameters
const int NISP = 0;     // NumberOfIntegerSharedParameters
const int NE   = 0;     // NumberOfEvents
const int NA   = 1;     // NumberOfAccessories
const int NIA  = 0;     // NumberOfIntegerAccessories
const int NDO  = 0;     // NumberOfPointsOfDenseOutput

void Linspace(vector<PRECISION>&, PRECISION, PRECISION, int);
void Logspace(vector<PRECISION>&, PRECISION, PRECISION, int);
void FillSolverObject(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>&, const vector<PRECISION>&, int);

int main()
{
	int BlockSize = 64;
	
	vector<PRECISION> Frequency(NT,0);
	Logspace(Frequency, 20.0, 1000.0, NT);
	
	// Setup CUDA a device
	ListCUDADevices();
	
	int MajorRevision  = 3;
	int MinorRevision  = 5;
	int SelectedDevice = SelectDeviceByClosestRevision(MajorRevision, MinorRevision);
	
	PrintPropertiesOfSpecificDevice(SelectedDevice);
	
	// Setup Solver
	ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> ScanKellerMiksis(SelectedDevice);
	
	ScanKellerMiksis.SolverOption(ThreadsPerBlock, BlockSize);
	ScanKellerMiksis.SolverOption(RelativeTolerance, 0, 1e-10);
	ScanKellerMiksis.SolverOption(RelativeTolerance, 1, 1e-10);
	ScanKellerMiksis.SolverOption(AbsoluteTolerance, 0, 1e-10);
	ScanKellerMiksis.SolverOption(AbsoluteTolerance, 1, 1e-10);
	
	
	// Simulation
	vector<PRECISION> GlobalMaxima(NT,0);
	FillSolverObject(ScanKellerMiksis, Frequency, NT);
	
	clock_t SimulationStart = clock();
	ScanKellerMiksis.SynchroniseFromHostToDevice(All);
	for (int i=0; i<1024; i++)
	{
		ScanKellerMiksis.Solve();
		ScanKellerMiksis.InsertSynchronisationPoint();
		ScanKellerMiksis.SynchroniseSolver();
	}
	ScanKellerMiksis.SynchroniseFromDeviceToHost(All);
	ScanKellerMiksis.InsertSynchronisationPoint();
	ScanKellerMiksis.SynchroniseSolver();
	clock_t TransientSimulationEnd = clock();
		cout << "Transient simulation time: " << (PRECISION)(TransientSimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "s" << endl << endl;
	
	for (int i=0; i<64; i++)
	{
		ScanKellerMiksis.Solve();
		ScanKellerMiksis.SynchroniseFromDeviceToHost(Accessories);
		ScanKellerMiksis.InsertSynchronisationPoint();
		ScanKellerMiksis.SynchroniseSolver();
		
		for (int tid=0; tid<NT; tid++)
			GlobalMaxima[tid] = fmax( ScanKellerMiksis.GetHost<PRECISION>(tid, Accessories, 0), GlobalMaxima[tid] );
	}
	
	// Save collected data to file
	ofstream DataFile;
	DataFile.open ( "KellerMiksis.txt" );
	int Width = 18;
	DataFile.precision(10);
	DataFile.flags(ios::scientific);
	
	for (int tid=0; tid<NT; tid++)
	{
		DataFile.width(8); DataFile << tid << ',';
		DataFile.width(Width); DataFile << Frequency[tid] << ',';
		DataFile.width(Width); DataFile << ScanKellerMiksis.GetHost<PRECISION>(tid, ActualState, 0) << ',';
		DataFile.width(Width); DataFile << ScanKellerMiksis.GetHost<PRECISION>(tid, ActualState, 1) << ',';
		DataFile.width(Width); DataFile << GlobalMaxima[tid];
		DataFile << '\n';
	}
	
	DataFile.close();
	clock_t TotalSimulationEnd = clock();
		cout << "Total simulation time: " << (PRECISION)(TotalSimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "s" << endl << endl;
	
	//ScanKellerMiksis.Print(DenseOutput, 25);
}

// ------------------------------------------------------------------------------------------------

void Linspace(vector<PRECISION>& x, PRECISION B, PRECISION E, int N)
{
    PRECISION Increment;
	
	x[0] = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}

void Logspace(vector<PRECISION>& x, PRECISION B, PRECISION E, int N)
{
    x[0] = B; 
	
	if ( N>1 )
	{
		x[N-1] = E;
		PRECISION ExpB = log10(B);
		PRECISION ExpE = log10(E);
		PRECISION ExpIncr = (ExpE-ExpB)/(N-1);
		for (int i=1; i<N-1; i++)
		{
			x[i] = pow(10,ExpB + i*ExpIncr);
		}
	}
}

// ------------------------------------------------------------------------------------------------

void FillSolverObject(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, const vector<PRECISION>& Values, int NumberOfThreads)
{
	// Declaration of physical control parameters
	PRECISION P1 = 1.5; // pressure amplitude1 [bar]
	PRECISION P2;       // frequency1          [kHz]
	PRECISION P3 = 0.0; // pressure amplitude2 [bar]
	PRECISION P4 = 0.0; // frequency2          [kHz]
	PRECISION P5 =  0.0; // phase shift          [-]
	PRECISION P6 = 10.0; // equilibrium radius   [mum]
	PRECISION P7 =  1.0; // ambient pressure     [bar]
	PRECISION P9 =  1.4; // polytrophic exponent [-]
	
	// Material properties
	PRECISION Pv  = 3.166775638952003e+03;
    PRECISION Rho = 9.970639504998557e+02;
    PRECISION ST  = 0.071977583160056;
    PRECISION Vis = 8.902125058209557e-04;
    PRECISION CL  = 1.497251785455527e+03;
	
	// Auxiliary variables
	PRECISION Pinf;
	PRECISION PA1;
	PRECISION PA2;
	PRECISION RE;
	PRECISION f1;
	PRECISION f2;
	
	for (int i=0; i<NumberOfThreads; i++)
	{	
		// Update physical parameters
		P2 = Values[i]; // frequency [kHz]
		
		Solver.SetHost(i, TimeDomain, 0, 0.0);
		Solver.SetHost(i, TimeDomain, 1, 1.0);
		
		// Initial conditions are the equilibrium condition y1=1; y2=0;
		Solver.SetHost(i, ActualState, 0, 1.0);
		Solver.SetHost(i, ActualState, 1, 0.0);
		
		// Scaling of physical parameters to SI
		Pinf = P7 * 1.0e5;
		PA1  = P1 * 1.0e5;
		PA2  = P3 * 1.0e5;
		RE   = P6 / 1.0e6;
		
		// Scale to angular frequency
		f1   = 2.0*PI*(P2*1000);
		f2   = 2.0*PI*(P4*1000);
		
		// System coefficients and other, auxiliary parameters
		Solver.SetHost(i, ControlParameters,  0, (2.0*ST/RE + Pinf - Pv) * pow(2.0*PI/RE/f1, 2.0) / Rho );
		Solver.SetHost(i, ControlParameters,  1, (1.0-3.0*P9) * (2*ST/RE + Pinf - Pv) * (2.0*PI/RE/f1) / CL/Rho );
		Solver.SetHost(i, ControlParameters,  2, (Pinf - Pv) * pow(2.0*PI/RE/f1, 2.0) / Rho );
		Solver.SetHost(i, ControlParameters,  3, (2.0*ST/RE/Rho) * pow(2.0*PI/RE/f1, 2.0) );
		Solver.SetHost(i, ControlParameters,  4, (4.0*Vis/Rho/pow(RE,2.0)) * (2.0*PI/f1) );
		Solver.SetHost(i, ControlParameters,  5, PA1 * pow(2.0*PI/RE/f1, 2.0) / Rho );
		Solver.SetHost(i, ControlParameters,  6, PA2 * pow(2.0*PI/RE/f1, 2.0) / Rho );
		Solver.SetHost(i, ControlParameters,  7, (RE*f1*PA1/Rho/CL) * pow(2.0*PI/RE/f1, 2.0) );
		Solver.SetHost(i, ControlParameters,  8, (RE*f2*PA2/Rho/CL) * pow(2.0*PI/RE/f1, 2.0) );
		Solver.SetHost(i, ControlParameters,  9, RE*f1/(2.0*PI)/CL );
		Solver.SetHost(i, ControlParameters, 10, 3.0*P9 );
		Solver.SetHost(i, ControlParameters, 11, P4/P2 );
		Solver.SetHost(i, ControlParameters, 12, P5 );
	}
}