#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <fstream>

#include "Lorenz_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

#define PI 3.14159265358979323846

using namespace std;

// Solver Configuration
#define SOLVER RK4        // RK4, RKCK45
#define PRECISION double  // float, double
const int NT   = 92160; // NumberOfThreads
const int SD   = 3;     // SystemDimension
const int NCP  = 1;     // NumberOfControlParameters
const int NSP  = 0;     // NumberOfSharedParameters
const int NISP = 0;     // NumberOfIntegerSharedParameters
const int NE   = 0;     // NumberOfEvents
const int NA   = 0;     // NumberOfAccessories
const int NIA  = 0;     // NumberOfIntegerAccessories
const int NDO  = 0;     // NumberOfPointsOfDenseOutput

void Linspace(vector<PRECISION>&, PRECISION, PRECISION, int);
void FillSolverObject(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>&, const vector<PRECISION>&, int);
void SaveData(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>&, int);

int main()
{
	int NumberOfProblems = NT;
	int BlockSize        = 64;
	
	ListCUDADevices();
	
	int MajorRevision  = 3;
	int MinorRevision  = 5;
	int SelectedDevice = SelectDeviceByClosestRevision(MajorRevision, MinorRevision);
	
	PrintPropertiesOfSpecificDevice(SelectedDevice);
	
	
	int NumberOfParameters_R = NumberOfProblems;
	PRECISION R_RangeLower = 0.0;
    PRECISION R_RangeUpper = 21.0;
		vector<PRECISION> Parameters_R_Values(NumberOfParameters_R,0);
		Linspace(Parameters_R_Values, R_RangeLower, R_RangeUpper, NumberOfParameters_R);
	
	
	ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> ScanLorenz(SelectedDevice);
	
	ScanLorenz.SolverOption(ThreadsPerBlock, BlockSize);
	ScanLorenz.SolverOption(InitialTimeStep, 1.0e-3);
	
	
	clock_t SimulationStart;
	clock_t SimulationEnd;
	
	FillSolverObject(ScanLorenz, Parameters_R_Values, NT);
	
	SimulationStart = clock();
	
	ScanLorenz.SynchroniseFromHostToDevice(All);
	
	ScanLorenz.Solve();
	
	ScanLorenz.SynchroniseFromDeviceToHost(All);
	ScanLorenz.InsertSynchronisationPoint();
	ScanLorenz.SynchroniseSolver();
	
	SimulationEnd = clock();
		cout << "Total simulation time:           " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl;
		cout << "Simulation time / 1000 RK4 step: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC / 10 << "ms" << endl;
		cout << "Ensemble size:                   " << NT << endl << endl;
		
	//SaveData(ScanLorenz, NT);
	
	cout << "Test finished!" << endl;
}

// AUXILIARY FUNCTION -----------------------------------------------------------------------------

void Linspace(vector<PRECISION>& x, PRECISION B, PRECISION E, int N)
{
    PRECISION Increment;
	
	x[0]   = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}

void FillSolverObject(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, const vector<PRECISION>& R_Values, int NumberOfThreads)
{
	int ProblemNumber = 0;
	for (int k=0; k<NumberOfThreads; k++)
	{
		Solver.SetHost(ProblemNumber, TimeDomain,  0, 0 );
		Solver.SetHost(ProblemNumber, TimeDomain,  1, 0.001*10000.0 );
		
		Solver.SetHost(ProblemNumber, ActualState, 0, 10.0 );
		Solver.SetHost(ProblemNumber, ActualState, 1, 10.0 );
		Solver.SetHost(ProblemNumber, ActualState, 2, 10.0 );
		
		Solver.SetHost(ProblemNumber, ControlParameters, 0, R_Values[k] );
		
		ProblemNumber++;
	}
}

void SaveData(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, int NumberOfThreads)
{
	ofstream DataFile;
	DataFile.open ( "Lorenz.txt" );
	
	int Width = 18;
	DataFile.precision(10);
	DataFile.flags(ios::scientific);
	
	for (int tid=0; tid<NumberOfThreads; tid++)
	{
		DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ControlParameters, 0) << ',';
		DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
		DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
		DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2);
		DataFile << '\n';
	}
	
	DataFile.close();
}